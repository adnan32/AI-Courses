//Task 1 - Odd-Even Sort using CUDA Adnan Altukleh & Abdulkarim Dawalibi

#include <vector>
#include <algorithm>
#include <iostream>
#include <chrono>
#include <hip/hip_runtime.h>
#define threads 1024

// Kernel function to add the elements of two arrays
__global__ void OES_kernel(int* data, int size) {
    int thread_offset = threadIdx.x * 2; 
    for (int i=1;i<=size;i++){
        for (int j = i % 2 + thread_offset;  j < size-1; j += threads *2) {
            if (data[j] > data[j + 1]) {
                int temp = data[j];
                data[j] = data[j + 1];
                data[j + 1] = temp;
            }
        }
        __syncthreads();
    }
}

void oddevensort(int* x, int size) {
    // Allocate GPU memory
    int *x_d;
    hipMalloc((void**) &x_d, size*sizeof(int));
    // Copy data to GPU memory
    hipMemcpy(x_d, x, size*sizeof(int), hipMemcpyHostToDevice);
    // Perform computation on GPU
    int numThreadsPerBlock = threads; 
    OES_kernel<<<1,numThreadsPerBlock>>>(x_d,size);
    hipDeviceSynchronize();// Wait for the GPU to finish
    // Copy data from GPU memory
    hipMemcpy(x, x_d, size*sizeof(int), hipMemcpyDeviceToHost);
    // Deallocate GPU memory
    hipFree(x_d);
}

void print_sort_status(std::vector<int> numbers)
{
    std::cout << "The input is sorted?: " << (std::is_sorted(numbers.begin(), numbers.end()) == 0 ? "False" : "True") << std::endl;
}

int main(){
    constexpr unsigned int size = 100000; // Number of elements in the input
    // Initialize a vector called numbers with integers of value 0 and size of size above
    std::vector<int> numbers(size);

    // Populate our vector with (pseudo)random numbers
    srand(static_cast<unsigned int>(time(0))); // Seed the random number generator
    generate(numbers.begin(), numbers.end(), rand);
    auto start = std::chrono::steady_clock::now();  // Start the timer
    // Sort using CUDA
    oddevensort(numbers.data(), numbers.size());
    // check if the vector is sorted
    auto end = std::chrono::steady_clock::now();  // End the timer
    print_sort_status(numbers);
    std::cout << "Elapsed time =  " << std::chrono::duration<double>(end - start).count() << " sec\n"; // Print the elapsed time
    return 0;
}