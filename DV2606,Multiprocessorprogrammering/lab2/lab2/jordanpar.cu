//Task 3 -Gaussjordan using CUDA Adnan Altukleh & Abdulkarim Dawalibi


#include <hip/hip_runtime.h>
#include <stdio.h>

#define MAX_SIZE 2048
#define NUM_THREADS 1024
#define PRNT 0
typedef double matrix[MAX_SIZE][MAX_SIZE];

int	N;		/* matrix size		*/
int	maxnum;		/* max number of element*/
char* Init;		/* matrix init type	*/
int	PRINT;		/* print switch		*/
matrix	A;		/* matrix A		*/
double	b[MAX_SIZE];	/* vector b             */
double	y[MAX_SIZE];	/* vector y             */

/* forward declarations */
void work(void);
__global__ void device_elimination1(double *deviceA,int N, double *b, double *y, int k);
__global__ void device_elimination2(double *deviceA,int N, double *y, int k);
__global__ void device_division(double *deviceA,int N, int k, double *y, double *b);
__global__ void add_device_elimination1(double *deviceA,int N, double *b, double *y,int k);
__global__ void add_device_elimination2(double *deviceA,int N, double *b, double *y,int k);
void Init_Matrix(void);
void Print_Matrix(void);
void Init_Default(void);
int Read_Options(int, char**);

int
main(int argc, char** argv)
{

    // int i, timestart, timeend, iter;
    
    Init_Default();		/* Init default values	*/
    Read_Options(argc, argv);	/* Read arguments	*/
    Init_Matrix();		/* Init the matrix	*/
    work();
    

    if (PRINT == 1)
        Print_Matrix();
}
//reach each row by index xN

__global__ void device_division(double *deviceA, int N, int k, double *y, double *b){


    for (int j = k + 1; j < N; j++){
            
        deviceA[(k*N)+j] = deviceA[(k*N)+j] / deviceA[(k*N)+k]; /* Division step */
            
    }
    y[k] = b[k] / deviceA[(k*N)+k];
    deviceA[(k*N)+k] = 1.0;

}



__global__ void device_elimination2(double *deviceA, int N, double *y, int k) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Ensure we don't go past the end of the matrix
    if (i < k) { // Ensuring that we don't modify the pivot row
        for (int j = k + 1; j < N; j++) {
            deviceA[i * N + j] -= deviceA[i * N + k] * deviceA[k * N + j];
        }
    }

}





__global__ void device_elimination1(double *deviceA,int N, double *b, double *y,int k)
{
    int thread_id = blockIdx.x*blockDim.x +threadIdx.x;
    int rows_per_thread = N / NUM_THREADS;
    int i, j;
    int start = thread_id * rows_per_thread; 
    int end = start + rows_per_thread; 

        for (i = start; i < end; i++) {
            
            if (i > k) {
            for (j = k + 1; j < N; j++){
                //((1= row number)* (N means jump one row)) + element index in the row
                deviceA[(i*N)+j] = deviceA[(i*N)+j] - deviceA[(i*N)+k] * deviceA[(k*N)+j];
            }
            }
        }

}

__global__ void add_device_elimination1(double *deviceA,int N, double *b, double *y,int k){

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i > k) {
        b[i] = b[i] - deviceA[(i*N)+k] * y[k];
        deviceA[(i*N)+k] = 0.0;
    }
}

__global__ void add_device_elimination2(double *deviceA,int N, double *b, double *y,int k){

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < k) {
        y[i] -= deviceA[i * N + k] * y[k];
        deviceA[i * N + k] = 0.0;
    }
}

void work(void)
{
    int  k;
    
    /* Gaussian elimination algorithm, Algo 8.4 from Grama */
    double *deviceA;
    double *deviceY;
    double *deviceB;

    hipMalloc(&deviceB,sizeof(double)*MAX_SIZE);
    hipMalloc(&deviceY,sizeof(double)*MAX_SIZE);
    hipMalloc(&deviceA,sizeof(matrix));

    hipMemcpy(deviceA,A,sizeof(matrix), hipMemcpyHostToDevice);
    hipMemcpy(deviceY,y,sizeof(double)*MAX_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(deviceB,b,sizeof(double)*MAX_SIZE, hipMemcpyHostToDevice);

    for (k = 0; k < N; k++) { /* Outer loop */   
        
        device_division<<<1,1>>>(deviceA,N,k,deviceY,deviceB);
        
        device_elimination1<<<128, NUM_THREADS>>>(deviceA,N,deviceB,deviceY,k);
        hipDeviceSynchronize();
        add_device_elimination1<<<2, NUM_THREADS>>>(deviceA,N,deviceB,deviceY,k);
        
        if(k>0){
            device_elimination2<<<2,NUM_THREADS>>>(deviceA,N,deviceY,k);
            hipDeviceSynchronize();
            add_device_elimination2<<<2,NUM_THREADS>>>(deviceA,N,deviceB,deviceY,k);
        }
        
    }
    
    hipMemcpy(A,deviceA,sizeof(matrix), hipMemcpyDeviceToHost);
    hipMemcpy(y,deviceY,sizeof(double)*MAX_SIZE, hipMemcpyDeviceToHost);
    hipFree(deviceA);
    hipFree(deviceB);
    hipFree(deviceY);

}

void
Init_Matrix()
{
    int i, j;

    printf("\nsize      = %dx%d ", N, N);
    printf("\nmaxnum    = %d \n", maxnum);
    printf("Init	  = %s \n", Init);
    printf("Initializing matrix...");

    if (strcmp(Init, "rand") == 0) {
        for (i = 0; i < N; i++) {
            for (j = 0; j < N; j++) {
                if (i == j) /* diagonal dominance */
                    A[i][j] = (double)(rand() % maxnum) + 5.0;
                else
                    A[i][j] = (double)(rand() % maxnum) + 1.0;
            }
        }
    }
    if (strcmp(Init, "fast") == 0) {
        for (i = 0; i < N; i++) {
            for (j = 0; j < N; j++) {
                if (i == j) /* diagonal dominance */
                    A[i][j] = 5.0;
                else
                    A[i][j] = 2.0;
            }
        }
    }

    /* Initialize vectors b and y */
    for (i = 0; i < N; i++) {
        b[i] = 2.0;
        y[i] = 1.0;
    }

    printf("done \n\n");
    if (PRINT == 1)
        Print_Matrix();
}

void
Print_Matrix()
{
    int i, j;

    printf("Matrix A:\n");
    for (i = 0; i < N; i++) {
        printf("[");
        for (j = 0; j < N; j++)
            printf(" %5.2f,", A[i][j]);
        printf("]\n");
    }
    printf("Vector y:\n[");
    for (j = 0; j < N; j++)
        printf(" %5.2f,", y[j]);
    printf("]\n");
    printf("\n\n");
}

void
Init_Default()
{
    N = MAX_SIZE;
    Init = "fast";
    maxnum = 15.0;
    PRINT = PRNT;
}

int
Read_Options(int argc, char** argv)
{
    char* prog;

    prog = *argv;
    while (++argv, --argc > 0)
        if (**argv == '-')
            switch (*++ * argv) {
            case 'n':
                --argc;
                N = atoi(*++argv);
                break;
            case 'h':
                printf("\nHELP: try sor -u \n\n");
                exit(0);
                break;
            case 'u':
                printf("\nUsage: gaussian [-n problemsize]\n");
                printf("           [-D] show default values \n");
                printf("           [-h] help \n");
                printf("           [-I init_type] fast/rand \n");
                printf("           [-m maxnum] max random no \n");
                printf("           [-P print_switch] 0/1 \n");
                exit(0);
                break;
            case 'D':
                printf("\nDefault:  n         = %d ", N);
                printf("\n          Init      = rand");
                printf("\n          maxnum    = 5 ");
                printf("\n          P         = 0 \n\n");
                exit(0);
                break;
            case 'I':
                --argc;
                Init = *++argv;
                break;
            case 'm':
                --argc;
                maxnum = atoi(*++argv);
                break;
            case 'P':
                --argc;
                PRINT = atoi(*++argv);
                break;
            default:
                printf("%s: ignored option: -%s\n", prog, *argv);
                printf("HELP: try %s -u \n\n", prog);
                break;
            }
}